#pragma once

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void test_kernel(int kernel_num, int m, int n, int k, float alpha, float *A, float *B, float beta, float *C, hipblasHandle_t handle);